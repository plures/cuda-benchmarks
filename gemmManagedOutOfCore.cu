#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "common.hh"

int main(int argc, char *argv[])
{
    size_t N = 16000;
    clock_t start, end; 
    hipblasHandle_t handle;
    float *a, *b, *c;
    const float alpha = 1;
    const float beta = 0;
    size_t count, nn;

    if (argc == 2) {
        N = checked_strtosize(argv[1]);
    }
    nn = checked_mul(N, N);
    count = checked_mul(nn, sizeof(float));

    check(hipblasCreate(&handle));

    start = clock();
    check(hipMallocManaged(&a, count));
    check(hipMallocManaged(&b, count));
    check(hipMallocManaged(&c, count));

    for (size_t i = 0; i < N*N; i++) {
        a[i] = i / 37.0;
        b[i] = i / 101.0;
    }
    end = clock();
    log("host: hipMallocManaged+init", start, end);

    start = clock();
    check(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                      N, N, N,
                      &alpha,
                      a, N,
                      b, N,
                      &beta,
                      c, N));
    check(hipDeviceSynchronize());
    end = clock();
    log("hipblasSgemm", start, end);

    return 0;
}
