#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "common.hh"

int main(int argc, char *argv[])
{
    size_t N = DEFAULT_N;
    if (argc==2) N = (size_t)atoi(argv[1]);
    printf("N=%zd\n", N);

    clock_t start, end; 
    hipblasHandle_t handle;
    float *a, *b, *c;
    const float alpha = 1;
    const float beta = 0;

    check(hipblasCreate(&handle));

    start = clock();
    check(hipMallocManaged(&a, N*N * sizeof(float)));
    check(hipMallocManaged(&b, N*N * sizeof(float)));
    check(hipMallocManaged(&c, N*N * sizeof(float)));

    for (size_t i = 0; i < N*N; i++) {
        a[i] = i / 37.0;
        b[i] = i / 101.0;
    }
    end = clock();
    log("host: hipMallocManaged+init", start, end);

    start = clock();
    check(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                      N, N, N,
                      &alpha,
                      a, N,
                      b, N,
                      &beta,
                      c, N));
    check(hipDeviceSynchronize());
    end = clock();
    log("hipblasSgemm", start, end);

    return 0;
}
