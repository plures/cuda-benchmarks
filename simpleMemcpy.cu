#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cinttypes>
#include <hip/hip_runtime.h>
#include "common.hh"


static __global__ void
f(const uint64_t a[], const uint64_t b[], uint64_t c[], int64_t N)
{
    int64_t index = threadIdx.x + blockIdx.x * blockDim.x;
    int64_t stride = blockDim.x * gridDim.x;

    for (int64_t i = index; i < N; i += stride) {
        c[i] = a[i] * b[i];
    }
}

static void
doit(const uint64_t a[], const uint64_t b[], uint64_t c[], int64_t N)
{
    int blockSize = 256;
    int64_t numBlocks = (N + blockSize - 1) / blockSize;

    f<<<numBlocks, blockSize>>>(a, b, c, N);
}

int
main(int argc, char *argv[])
{
    size_t N = DEFAULT_N;
    if (argc==2) N = (size_t)atoi(argv[1]);
    printf("N=%zd\n", N);

    clock_t start_program, end_program;
    clock_t start, end;
    uint64_t *a, *b, *c;
    uint64_t *da, *db, *dc;
    size_t count = N * sizeof(uint64_t);

    start_program = clock();

    start = clock();
    check(hipHostMalloc(&a, count));
    check(hipHostMalloc(&b, count));
    check(hipHostMalloc(&c, count));
    end = clock();
    log("host: MallocHost", start, end);

    start = clock();
    for (size_t i = 0; i < N; i++) {
        a[i] = 3;
        b[i] = 5;
    }
    end = clock();
    log("host: init arrays", start, end);

    start = clock();
    check(hipMalloc(&da, count));
    check(hipMalloc(&db, count));
    check(hipMalloc(&dc, count));

    check(hipMemcpy(da, a, count, hipMemcpyHostToDevice));
    check(hipMemcpy(db, b, count, hipMemcpyHostToDevice));

    doit(da, db, dc, N);

    check(hipMemcpy(c, dc, count, hipMemcpyDeviceToHost));

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    end = clock();
    log("device: malloc+copy+compute", start, end);

    start = clock();
    for (size_t i = 0; i < N; i++) {
        if (a[i] != 3 || b[i] != 5 || c[i] != 15) {
            fprintf(stderr, "unexpected result a: %lu  b: %lu  c: %lu\n",
                    a[i], b[i], c[i]);
            exit(1);
        }
    }
    end = clock();
    log("host: access all arrays", start, end);

    start = clock();
    for (size_t i = 0; i < N; i++) {
        if (a[i] != 3 || b[i] != 5 || c[i] != 15) {
            fprintf(stderr, "unexpected result a: %lu  b: %lu  c: %lu\n",
                    a[i], b[i], c[i]);
            exit(1);
        }
    }
    end = clock();
    log("host: access all arrays a second time", start, end);

    start = clock();
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
    end = clock();
    log("host: free", start, end);

    end_program = clock();
    log("total", start_program, end_program);

    return 0;
}
