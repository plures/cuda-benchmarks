#include "hip/hip_runtime.h"
#include <cstdio>
#include <cinttypes>
#include <hip/hip_runtime.h>
#include "common.hh"


const size_t N = 500000000;


static __global__ void
f(const uint64_t a[], const uint64_t b[], uint64_t c[], int64_t N)
{
    int64_t index = threadIdx.x + blockIdx.x * blockDim.x;
    int64_t stride = blockDim.x * gridDim.x;

    for (int64_t i = index; i < N; i += stride) {
        c[i] = a[i] * b[i];
    }
}

static void
doit(const uint64_t a[], const uint64_t b[], uint64_t c[], int64_t N)
{
    int blockSize = 256;
    int64_t numBlocks = (N + blockSize - 1) / blockSize;

    f<<<numBlocks, blockSize>>>(a, b, c, N);
}

int
main(void)
{
    clock_t start_program, end_program;
    clock_t start, end;
    uint64_t *a, *b, *c;
    uint64_t *da, *db, *dc;
    size_t count = N * sizeof(uint64_t);

    start_program = clock();

    start = clock();
    check(hipHostMalloc(&a, count));
    check(hipHostMalloc(&b, count));
    check(hipHostMalloc(&c, count));
    end = clock();
    log("host: malloc", start, end);

    start = clock();
    for (size_t i = 0; i < N; i++) {
        a[i] = 3;
        b[i] = 5;
    }
    end = clock();
    log("host: init arrays", start, end);

    start = clock();
    check(hipMalloc(&da, count));
    check(hipMalloc(&db, count));
    check(hipMalloc(&dc, count));

    check(hipMemcpy(da, a, count, hipMemcpyHostToDevice));
    check(hipMemcpy(db, b, count, hipMemcpyHostToDevice));
    check(hipMemcpy(dc, c, count, hipMemcpyHostToDevice));

    doit(da, db, dc, N);

    check(hipMemcpy(c, dc, count, hipMemcpyDeviceToHost));

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    end = clock();
    log("device: malloc+copy+compute", start, end);

    start = clock();
    for (size_t i = 0; i < N; i++) {
        if (a[i] != 3 || b[i] != 5 || c[i] != 15) {
            fprintf(stderr, "unexpected result a: %lu  b: %lu  c: %lu\n",
                    a[i], b[i], c[i]);
            exit(1);
        }
    }
    end = clock();
    log("host: access all arrays", start, end);

    start = clock();
    for (size_t i = 0; i < N; i++) {
        if (a[i] != 3 || b[i] != 5 || c[i] != 15) {
            fprintf(stderr, "unexpected result a: %lu  b: %lu  c: %lu\n",
                    a[i], b[i], c[i]);
            exit(1);
        }
    }
    end = clock();
    log("host: access all arrays a second time", start, end);

    start = clock();
    hipFree(a);
    hipFree(b);
    hipFree(c);
    end = clock();
    log("host: free", start, end);

    end_program = clock();
    log("total", start_program, end_program);

    return 0;
}
