#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cublasXt.h>
#include "common.hh"


int
main(int argc, char *argv[])
{
    size_t N = 16000;
    clock_t start, end; 
    cublasXtHandle_t handle;
    int devices[1] = {0};
    float *a, *b, *c;
    const float alpha = 1;
    const float beta = 0;
    size_t count, nn;

    if (argc == 2) {
        N = checked_strtosize(argv[1]);
    }
    nn = checked_mul(N, N);
    count = checked_mul(nn, sizeof(float));

    check(cublasXtCreate(&handle));
    check(cublasXtDeviceSelect(handle, 1, devices));

    start = clock();
    check(hipHostMalloc(&a, count));
    check(hipHostMalloc(&b, count));
    check(hipHostMalloc(&c, count));

    for (size_t i = 0; i < N*N; i++) {
        a[i] = i / 37.0;
        b[i] = i / 101.0;
    }
    end = clock();
    log("host: MallocHost+init", start, end);

    start = clock();
    check(cublasXtSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                        N, N, N,
                        &alpha,
                        a, N,
                        b, N,
                        &beta,
                        c, N));
    end = clock();
    log("cublasXtSgemm", start, end);

    return 0;
}
